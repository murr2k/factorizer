#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    std::cout << "Testing CUDA device detection..." << std::endl;
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    std::cout << "hipGetDeviceCount returned: " << hipGetErrorString(err) << std::endl;
    std::cout << "Device count: " << deviceCount << std::endl;
    
    if (err == hipSuccess && deviceCount > 0) {
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t prop;
            hipGetDeviceProperties(&prop, i);
            std::cout << "\nDevice " << i << ": " << prop.name << std::endl;
            std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
            std::cout << "  Total memory: " << prop.totalGlobalMem / (1024*1024) << " MB" << std::endl;
        }
    }
    
    // Try to set device
    if (deviceCount > 0) {
        err = hipSetDevice(0);
        std::cout << "\ncudaSetDevice(0) returned: " << hipGetErrorString(err) << std::endl;
    }
    
    return 0;
}
#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    std::cout << "Minimal CUDA test" << std::endl;
    
    // Force runtime init
    hipFree(0);
    
    // Check devices
    int count = 0;
    hipError_t err = hipGetDeviceCount(&count);
    
    std::cout << "Device count: " << count << std::endl;
    std::cout << "Error: " << hipGetErrorString(err) << std::endl;
    
    if (count > 0) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        std::cout << "GPU: " << prop.name << std::endl;
    }
    
    return 0;
}
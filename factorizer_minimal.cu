#include "hip/hip_runtime.h"
/**
 * Minimal 128-bit Factorizer
 * Testing without cuRAND to isolate issues
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <time.h>

#include "uint128_improved.cuh"
#include "barrett_reduction.cuh"

#define MAX_FACTORS 32

// Simple Pollard's Rho without cuRAND
__global__ void pollards_rho_simple(
    uint128_t n,
    uint128_t* factors,
    int* factor_count,
    int max_iterations
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Simple deterministic start based on thread ID
    uint128_t x((tid + 2) % n.low, 0);
    uint128_t y = x;
    uint128_t c(1 + (tid % 10), 0);
    
    // Barrett reduction setup
    Barrett128 barrett;
    barrett.n = n;
    barrett.precompute();
    
    uint128_t factor(1, 0);
    
    for (int i = 0; i < max_iterations && factor.low == 1; i++) {
        // x = (x^2 + c) mod n
        uint256_t x_squared = multiply_128_128(x, x);
        x = barrett.reduce(x_squared.low_128());
        x = add_128(x, c);
        if (x >= n) x = subtract_128(x, n);
        
        // y = f(f(y))
        for (int j = 0; j < 2; j++) {
            uint256_t y_squared = multiply_128_128(y, y);
            y = barrett.reduce(y_squared.low_128());
            y = add_128(y, c);
            if (y >= n) y = subtract_128(y, n);
        }
        
        // Calculate GCD
        uint128_t diff = (x > y) ? subtract_128(x, y) : subtract_128(y, x);
        factor = gcd_128(diff, n);
        
        // Check if found non-trivial factor
        if (factor.low > 1 && factor < n) {
            int idx = atomicAdd(factor_count, 1);
            if (idx < MAX_FACTORS) {
                factors[idx] = factor;
            }
            break;
        }
    }
}

// Helper to convert string to uint128_t
uint128_t string_to_uint128(const char* str) {
    uint128_t result(0, 0);
    
    for (int i = 0; str[i] != '\0'; i++) {
        // result = result * 10 + digit
        uint128_t ten(10, 0);
        uint256_t prod = multiply_128_128(result, ten);
        result = prod.low_128();
        result = add_128(result, uint128_t(str[i] - '0', 0));
    }
    
    return result;
}

// Main function
int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <number>\n", argv[0]);
        return 1;
    }
    
    // Convert input
    uint128_t n = string_to_uint128(argv[1]);
    printf("Factoring: %s\n", argv[1]);
    
    // Allocate device memory
    uint128_t* d_factors;
    int* d_factor_count;
    hipMalloc(&d_factors, MAX_FACTORS * sizeof(uint128_t));
    hipMalloc(&d_factor_count, sizeof(int));
    hipMemset(d_factor_count, 0, sizeof(int));
    
    // Start timing
    clock_t start = clock();
    
    // Run Pollard's Rho
    pollards_rho_simple<<<32, 256>>>(n, d_factors, d_factor_count, 100000);
    hipDeviceSynchronize();
    
    // Get results
    int factor_count;
    hipMemcpy(&factor_count, d_factor_count, sizeof(int), hipMemcpyDeviceToHost);
    
    if (factor_count > 0) {
        uint128_t* h_factors = new uint128_t[MAX_FACTORS];
        hipMemcpy(h_factors, d_factors, MAX_FACTORS * sizeof(uint128_t), hipMemcpyDeviceToHost);
        
        printf("Found %d factor(s) in %.3f seconds:\n", 
               factor_count, (double)(clock() - start) / CLOCKS_PER_SEC);
        
        for (int i = 0; i < factor_count && i < MAX_FACTORS; i++) {
            printf("  %llu\n", h_factors[i].low);
        }
        
        // Verify
        if (factor_count >= 1) {
            uint128_t cofactor = divide_128_64(n, h_factors[0].low);
            printf("Factorization: %llu × %llu = %s\n", 
                   h_factors[0].low, cofactor.low, argv[1]);
        }
        
        delete[] h_factors;
    } else {
        printf("No factors found in %.3f seconds\n", 
               (double)(clock() - start) / CLOCKS_PER_SEC);
    }
    
    // Cleanup
    hipFree(d_factors);
    hipFree(d_factor_count);
    
    return 0;
}
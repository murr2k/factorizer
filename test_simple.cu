#include "hip/hip_runtime.h"
/**
 * Ultra-simple CUDA test
 */

#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void simple_kernel(int* result) {
    *result = 42;
}

int main() {
    printf("Simple CUDA Test\n");
    printf("================\n");
    
    // Check CUDA
    int device_count;
    hipError_t err = hipGetDeviceCount(&device_count);
    if (err \!= hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }
    
    printf("CUDA devices found: %d\n", device_count);
    
    // Simple kernel test
    int* d_result;
    hipMalloc(&d_result, sizeof(int));
    
    simple_kernel<<<1, 1>>>(d_result);
    
    int h_result;
    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);
    
    printf("Kernel result: %d (expected 42)\n", h_result);
    
    hipFree(d_result);
    
    printf("Test completed successfully\!\n");
    
    return 0;
}
EOF < /dev/null

#include <hip/hip_runtime.h>
#include <stdio.h>

// Simple kernel
__global__ void testKernel() {
    printf("Hello from GPU thread %d\n", threadIdx.x);
}

int main() {
    printf("CUDA Runtime Test\n");
    
    // Initialize runtime
    hipError_t err = hipFree(0);
    printf("cudaFree(0): %s\n", hipGetErrorString(err));
    
    // Get device count
    int deviceCount;
    err = hipGetDeviceCount(&deviceCount);
    printf("cudaGetDeviceCount: %s, count=%d\n", hipGetErrorString(err), deviceCount);
    
    if (deviceCount > 0) {
        // Launch a simple kernel
        testKernel<<<1, 4>>>();
        err = hipDeviceSynchronize();
        printf("Kernel launch: %s\n", hipGetErrorString(err));
    }
    
    return 0;
}